#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <iostream> 
 
__global__ void addKernel(float **C,  float **A, int RR, int CC)     //������
{
	//threadIdx(.x/.y/.z����ά����)���߳�����block�и���ά���ϵ��̺߳�
	//blockDim.x�������x���ϵ��߳�������blockDim.y�������y���ϵ��߳�������blockDim.z�������z���ϵ��߳�����
	//blockIdx(.x/.y/.z����ά����)��������grid�и���ά���ϵĿ��

	int idx = threadIdx.x + blockDim.x * blockIdx.x; 
	int idy = threadIdx.y + blockDim.y * blockIdx.y;	

	float tem;
	int m;
	int n;
	int win;
	win=5;

	//�Ծ���A���о�ֵ�˲�
	if (idx < CC-win && idy < RR-win && idx>=win && idy>=win)
	{
	  tem=0;
	  for (  m =-win; m <= win ; m++)
	   		for ( n = -win; n <= win; n++)
				{
					tem = tem + A[idy+m][idx+n];
				}
	//idx idy���Կ��������꣬�����ڸ�if��Χ������������ɨ�裬���￴��ÿ������ȡһ����
			C[idy][idx] =  tem/(2*win+1)/(2*win+1);

	}
}
__global__ void addKernel2(float **C, float **A, int RR, int CC)     //������
{
	//threadIdx(.x/.y/.z����ά����)���߳�����block�и���ά���ϵ��̺߳�
	//blockDim.x�������x���ϵ��߳�������blockDim.y�������y���ϵ��߳�������blockDim.z�������z���ϵ��߳�����
	//blockIdx(.x/.y/.z����ά����)��������grid�и���ά���ϵĿ��
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int idy = threadIdx.y + blockDim.y * blockIdx.y;
	float temp;
	int m;
	int n;
	int win;
	win = 5;
	float value[9];

	//�Ծ���A������ֵ�˲�
	if (idx < CC - win && idy < RR - win && idx >= win && idy >= win)
	{
		value[0] = A[idy - 1][idx - 1];
		value[1] = A[idy - 1][idx];
		value[2] = A[idy - 1][idx + 1];
		value[3] = A[idy][idx - 1];
		value[4] = A[idy][idx];
		value[5] = A[idy][idx + 1];
		value[6] = A[idy + 1][idx - 1];
		value[7] = A[idy + 1][idx];
		value[8] = A[idy + 1][idx + 1];
		for (m = 0; m < 9; m++)
		{
			for (n = 0; n < 9 - m - 1; n++)
			{
				if (value[n] > value[n + 1])
				{
					temp = value[n];
					value[n] = value[n + 1];
					value[n + 1] = temp;
				}
			}
		}
		//idx idy���Կ��������꣬�����ڸ�if��Χ������������ɨ�裬���￴��ÿ������ȡһ����
		C[idy][idx] = value[4];
	}
}



void main()
{	
	//��¼��ʼʱ��
	clock_t start, finish;
	float costtime;
	start = clock();	
				
    //�����������ֵ
	int Row, Col, a, b;
	Row= 1024;
	Col= 1024;
	a = 16;
	b = 16;
	//�������������CPU�ڴ�
	float **A = (float **)malloc(sizeof(float*) * Row);
	float **C = (float **)malloc(sizeof(float*) * Row);
	float *dataA = (float *)malloc(sizeof(float) * Row * Col);
	float *dataC = (float *)malloc(sizeof(float) * Row * Col);


	//�������
	float **d_A;
	float **d_C;
	float *d_dataA;
	float *d_dataC;

	float *temp,*temp1;
	FILE *ff;
	FILE *fid;

	
	//����GPU�ڴ�
	hipMalloc((void**)&d_A, sizeof(float **) * Row);
	hipMalloc((void**)&d_C, sizeof(float **) * Row);
	hipMalloc((void**)&d_dataA, sizeof(float) *Row*Col);
	hipMalloc((void**)&d_dataC, sizeof(float) *Row*Col);

	//����CPU�ڴ�
	temp = (float *)calloc(Row * Col, sizeof(float));
	temp1 = (float *)calloc(Col, sizeof(float));

	//��ʼ������
    for (int i=0;i<Row * Col;i++)
		 temp[i]=0;

	//���ı�����
	fid = fopen("C:\\Users\\LENOVO\\Desktop\\Cuda������2\\Meanfiltering\\11\\inputimage4", "rb");
	fread(temp, sizeof(float), Row * Col, fid);

	//���ı���ֵ����dataA
	for (int i = 0; i<Row * Col; i++)
	{
		dataA[i]= temp[i];
	}
    fclose(fid);


	//������ָ��Aָ���豸����λ�ã�Ŀ�������豸����ָ���ܹ�ָ���豸����һ��ָ��
	//A��dataA ���������豸�ϣ��������ߵĶ�Ӧ��ϵ
	for (int i = 0; i < Row; i++)
	 {
		A[i] = d_dataA + Col * i;
		C[i] = d_dataC + Col * i;
	}
	
	//��CPU�������ϴ���GPU��
	hipMemcpy(d_A, A, sizeof(float*) * Row, hipMemcpyHostToDevice);
	hipMemcpy(d_C, C, sizeof(float*) * Row, hipMemcpyHostToDevice);
	hipMemcpy(d_dataA, dataA, sizeof(float) * Row * Col, hipMemcpyHostToDevice);

	//ȷ��grid��block��thread�Ĵ�С
	dim3 threadPerBlock(a,b); 
	dim3 blockNumber( (Col + threadPerBlock.x - 1)/ threadPerBlock.x, (Row + threadPerBlock.y - 1) / threadPerBlock.y );
	printf("Block(%d,%d)   Grid(%d,%d).\n", threadPerBlock.x, threadPerBlock.y, blockNumber.x, blockNumber.y);

	//��d_C��d_A��Row��Col��ֵ���������������
	//addKernel << <blockNumber, threadPerBlock >> > (d_C, d_A, Row, Col);
	addKernel2 << <blockNumber, threadPerBlock >> > (d_C, d_A, Row, Col);
	
	//������������-һ������ָ��
	hipMemcpy(dataC, d_dataC, sizeof(float) * Row * Col, hipMemcpyDeviceToHost);


	//������õ�����д���ı���ʽ���
	ff = fopen("C:\\Users\\LENOVO\\Desktop\\Cuda������2\\Meanfiltering\\11\\midd2.dat", "wb");
	for (int i = 0; i < Row; i++)
	{
		for (int j = 0;j < Col;j++)
		{
			temp1[j] = dataC[i * Col +j];
		}
		fwrite(temp1, sizeof(float), Col, ff);

	}
	fclose(ff);


	//��ý���ʱ��,����ʾ��ʱ���
	finish = clock();         
	costtime = (float)(finish - start) / CLOCKS_PER_SEC; 
	printf("1024*1024ͼ���ֵ�˲�GPU����Time=(%3.6f s)\n",costtime);

	//�ͷ�CPU��GPU�ڴ�
	free(A);
	free(C);
	free(dataA);
	free(dataC);
	hipFree(d_A);
	hipFree(d_C);
	hipFree(d_dataA);
	hipFree(d_dataC);

	getchar();
}
